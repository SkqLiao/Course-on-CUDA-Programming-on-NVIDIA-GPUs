#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime_api.h"

///////////////////////////////////////////////////////////////////////////////
// CPU routine
///////////////////////////////////////////////////////////////////////////////

void scan_gold(double *odata, double *idata, const unsigned int len) {
  odata[0] = 0;
  for (int i = 1; i < len; i++) odata[i] = idata[i - 1] + odata[i - 1];
}

///////////////////////////////////////////////////////////////////////////////
// GPU routine
///////////////////////////////////////////////////////////////////////////////

__device__ volatile int current_block = 0;
__device__ volatile double current_sum = 0.0f;

__global__ void scan(int N, double *g_odata, double *g_idata) {
  // Dynamically allocated shared memory for scan kernels

  extern __shared__ double tmp[];

  double temp;
  int tid = threadIdx.x;
  int rid = tid + blockDim.x * blockIdx.x;

  if (rid >= N) return;
  // read input into shared memory

  temp = g_idata[rid];
  tmp[tid] = temp;

  // scan up the tree

  for (int d = 1; d < blockDim.x; d = 2 * d) {
    __syncthreads();

    if (tid - d >= 0) temp = temp + tmp[tid - d];

    __syncthreads();

    tmp[tid] = temp;
  }

  // write results to global memory

  __syncthreads();

  temp = tmp[tid];

  __syncthreads();

  do {
  } while (current_block < blockIdx.x);
  temp += current_sum;
  __threadfence();
  if (tid == blockDim.x - 1) {
    current_sum += tmp[blockDim.x - 1];
    current_block++;
  }

  if (rid < N) g_odata[rid + 1] = temp;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv) {
  int num_elements, num_threads, num_blocks, mem_size, shared_mem_size;

  double *h_data, *reference;
  double *d_idata, *d_odata;

  // initialise card

  findCudaDevice(argc, argv);

  num_elements = 100000000;
  num_threads = 1024;
  num_blocks = (num_elements + num_threads - 1) / num_threads;
  mem_size = sizeof(double) * num_elements;

  // allocate host memory to store the input data
  // and initialize to integer values between 0 and 1000

  h_data = (double *)malloc(mem_size);

  for (int i = 0; i < num_elements; i++)
    h_data[i] = floorf(1000 * (rand() / (double)RAND_MAX));

  // compute reference solution

  reference = (double *)malloc(mem_size);
  scan_gold(reference, h_data, num_elements);

  // allocate device memory input and output arrays

  checkCudaErrors(hipMalloc((void **)&d_idata, mem_size));
  checkCudaErrors(hipMalloc((void **)&d_odata, mem_size));

  // copy host memory to device input array

  checkCudaErrors(
      hipMemcpy(d_idata, h_data, mem_size, hipMemcpyHostToDevice));

  // execute the kernel

  shared_mem_size = sizeof(double) * num_threads;

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  scan<<<num_blocks, num_threads, shared_mem_size>>>(num_elements, d_odata,
                                                     d_idata);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("\nscan2: %.1f (ms) \n", milli);

  getLastCudaError("scan kernel execution failed");

  // copy result from device to host

  checkCudaErrors(
      hipMemcpy(h_data, d_odata, mem_size, hipMemcpyDeviceToHost));

  // check results

  double err = 0.0;
  for (int i = 0; i < num_elements; i++) {
    err += (h_data[i] - reference[i]) * (h_data[i] - reference[i]);
    // printf("%d %f %f \n", i, h_data[i], reference[i]);
  }
  printf("rms scan error  = %f\n", sqrt(err / num_elements));

  // cleanup memory

  free(h_data);
  free(reference);
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
