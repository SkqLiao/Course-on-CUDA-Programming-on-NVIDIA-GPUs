//
// include files
//


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//
// template kernel routine
//

template <class T>
__global__ void my_first_kernel(T *x) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  x[tid] = threadIdx.x;
}

//
// CUDA routine to be called by main code
//

extern int prac6(int nblocks, int nthreads) {
  float *h_x, *d_x;
  int *h_i, *d_i;
  double *h_y, *d_y;
  int nsize, n;

  // allocate memory for arrays

  nsize = nblocks * nthreads;

  h_x = (float *)malloc(nsize * sizeof(float));
  hipMalloc((void **)&d_x, nsize * sizeof(float));

  h_i = (int *)malloc(nsize * sizeof(int));
  hipMalloc((void **)&d_i, nsize * sizeof(int));

  h_y = (double *)malloc(nsize * sizeof(double));
  hipMalloc((void **)&d_y, nsize * sizeof(double));

  // execute kernel for float

  my_first_kernel<<<nblocks, nthreads>>>(d_x);
  hipMemcpy(h_x, d_x, nsize * sizeof(float), hipMemcpyDeviceToHost);
  for (n = 0; n < nsize; n++) printf(" n,  x  =  %d  %f \n", n, h_x[n]);

  // execute kernel for ints

  my_first_kernel<<<nblocks, nthreads>>>(d_i);
  hipMemcpy(h_i, d_i, nsize * sizeof(int), hipMemcpyDeviceToHost);
  for (n = 0; n < nsize; n++) printf(" n,  i  =  %d  %d \n", n, h_i[n]);

  my_first_kernel<<<nblocks, nthreads>>>(d_y);
  hipMemcpy(h_y, d_y, nsize * sizeof(double), hipMemcpyDeviceToHost);
  for (n = 0; n < nsize; n++) printf(" n,  i  =  %d  %lf \n", n, h_y[n]);

  // free memory

  hipFree(d_x);
  free(h_x);
  hipFree(d_i);
  free(h_i);

  return 0;
}
